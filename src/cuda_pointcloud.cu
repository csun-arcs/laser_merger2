#include "hip/hip_runtime.h"
#include <laser_merger2/cuda_pointcloud.h>
#include <iostream>
#include <hipblas.h>

#define THREADS_PER_BLOCK (1024)

// Error checking macro
#define CHECK_CUBLAS_ERROR(err) { \
    if (err != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "CUBLAS error: " << err << std::endl; \
        exit(1); \
    } \
}

static hipblasHandle_t handle;
static hipblasStatus_t status;

void InitGPUDev(void)
{
    status = hipblasCreate(&handle);
    CHECK_CUBLAS_ERROR(status);
    printf("CUBLAS OPENED\n");
}

__global__ void TransformArrayLaserLink(float* point, float minAng, float maxAng, int size)
{
    int idx = blockIdx.x * blockDim.x + threadIdx.x;

    if (idx < size)
    {
        float angIncrement = (maxAng - minAng) / size;
        float angle = minAng + idx * angIncrement;
        float dist = point[idx];
        __sincosf(angle, &point[idx + size], &point[idx]); 
        point[idx] = dist * point[idx];
        point[idx + size] = dist * point[idx + size];
        //printf("dist = %f, angle = %f, x = %f, y = %f\n, z = %f, uint = %f\n", dist, angle, point[idx], point[idx + size], point[idx + 2 * size], point[idx + 3 * size]);
    }
}

void InitGpuBuffer(float **dist, float **transformTF, float **outPoint, int size)
{
    hipMallocManaged(dist, 4 * size * sizeof(float));
    hipMallocManaged(transformTF, 16 * sizeof(float));
    hipMallocManaged(outPoint, 4 * size * sizeof(float));
}

void ScanToTransformedPointCloud(float *point, float minAng, float maxAng, int dataSize, float *transformMatrix, float *output)
{
    const float alpha = 1.0, beta = 0.0;
    int M = 4, N = dataSize, K = 4;

    //printf("minAng = %f, maxAng = %f\n", minAng, maxAng);

    int threadsPerBlock = THREADS_PER_BLOCK;
	int blocksPerGrid = (dataSize + threadsPerBlock - 1) / threadsPerBlock;

    TransformArrayLaserLink<<<blocksPerGrid, threadsPerBlock>>>(&point[0], minAng, maxAng, dataSize);
    hipDeviceSynchronize();

    /*for(int i = 0; i < dataSize; i++)
    {
        printf("x = %f, y = %f, z = %f, unit = %f\n", point[i], point[i + dataSize], point[i + dataSize * 2], point[i + dataSize * 3]);
    }*/

    /*for(int i = 0; i < 16; i++)
    {
        printf("tfMatrix[%d] = %f\n", i, transformMatrix[i]);
    }*/

    hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, M, K, &alpha, point, N, transformMatrix, K, &beta, output, N);
	hipDeviceSynchronize();

    /*for(int i = 0; i < dataSize; i++)
    {
        printf("x = %f, y = %f, z = %f, unit = %f\n", output[i], output[i + dataSize], output[i + dataSize * 2], output[i + dataSize * 3]);
    }*/
}